#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include "cuda_utils.h"
#include "timer.c"

typedef float dtype;
#define MAX_THREADS 256
#define MAX_BLOCKS 64
#define MIN(x,y) ((x < y) ? x : y)
#define TILE_SIZE    16
#define BLOCK_DIM  16

unsigned int nextPow2( unsigned int x ) {
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}


__global__ void matTrans(dtype* AT, dtype* A, int N, int threads, int blocks){
 	__shared__ dtype tile[TILE_SIZE][TILE_SIZE + 1];
  
  	int block_width = TILE_SIZE * gridDim.x;
  	int x,y;
  	x = TILE_SIZE * blockIdx.x + threadIdx.x;
  	y = TILE_SIZE * blockIdx.y + threadIdx.y;

  	for (int i = 0; i < TILE_SIZE; i += BLOCK_DIM)
 	{
     	tile[threadIdx.x+i][threadIdx.y] = A[(x+i)*block_width + y];
  	}
	__syncthreads();

  	x = blockIdx.y * TILE_SIZE + threadIdx.y;  

  	y = blockIdx.x * TILE_SIZE + threadIdx.x;

	for (int i = 0; i < TILE_SIZE; i += BLOCK_DIM)
	{
		AT[(x+i)*block_width + y] = tile[threadIdx.x][threadIdx.y + i];
	}

}

void
parseArg (int argc, char** argv, int* N)
{
	if(argc == 2) {
		*N = atoi (argv[1]);
		assert (*N > 0);
	} else {
		fprintf (stderr, "usage: %s <N>\n", argv[0]);
		exit (EXIT_FAILURE);
	}
}

void
initArr (dtype* in, int N)
{
	int i;

	for(i = 0; i < N; i++) {
		in[i] = (dtype) rand () / RAND_MAX;
	}
}

void
cpuTranspose (dtype* A, dtype* AT, int N)
{
	int i, j;
	for(i = 0; i < N; i++) {
		for(j = 0; j < N; j++) {
			AT[j * N + i] = A[i * N + j];
		}
	}
}

int
cmpArr (dtype* a, dtype* b, int N)
{
	int cnt, i;

	cnt = 0;
	for(i = 0; i < N; i++) {
		if(abs(a[i] - b[i]) > 1e-6) cnt++;
	}

	return cnt;
}

void
gpuTranspose (dtype* A, dtype* AT, int N)
{
  struct stopwatch_t* timer = NULL;
  long double t_gpu;
	
  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();
  int blocks;
  int threads; 
  threads = (N < MAX_THREADS) ? nextPow2(N) : MAX_BLOCKS;
  blocks = (N + threads - 1) / threads;

  dim3 gb(N/TILE_SIZE, N/TILE_SIZE, 1);
  dim3 tb(TILE_SIZE, BLOCK_DIM, 1);

  stopwatch_start (timer);

	matTrans <<<gb, tb>>> (AT, A, N, threads, blocks);

  hipDeviceSynchronize ();
  t_gpu = stopwatch_stop (timer);
  fprintf (stderr, "GPU transpose: %Lg secs ==> %Lg billion elements/second\n",
           t_gpu, (N * N) / t_gpu * 1e-9 );

}

int 
main(int argc, char** argv)
{


  /* variables */
	dtype *A, *ATgpu, *ATcpu;
  int err;

	int N;

  struct stopwatch_t* timer = NULL;
  long double t_cpu;

	N = -1;
	parseArg (argc, argv, &N);

  /* input and output matrices on host */
  /* output */
  ATcpu = (dtype*) malloc (N * N * sizeof (dtype));
  ATgpu = (dtype*) malloc (N * N * sizeof (dtype));

  /* input */
  A = (dtype*) malloc (N * N * sizeof (dtype));

	initArr (A, N * N);

	/* GPU transpose kernel */
	gpuTranspose (A, ATgpu, N);

  /* Setup timers */
  stopwatch_init ();
  timer = stopwatch_create ();

	stopwatch_start (timer);
  /* compute reference array */
	cpuTranspose (A, ATcpu, N);
  t_cpu = stopwatch_stop (timer);
  fprintf (stderr, "Time to execute CPU transpose kernel: %Lg secs\n",
           t_cpu);

  /* check correctness */
	err = cmpArr (ATgpu, ATcpu, N * N);
	if(err) {
		fprintf (stderr, "Transpose failed: %d\n", err);
	} else {
		fprintf (stderr, "Transpose successful\n");
	}

	free (A);
	free (ATgpu);
	free (ATcpu);

  return 0;
}
